#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceAd(int n, float* ptrResultGM);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void sliceAd(int n, float* ptrResultGM)
    {

    extern __shared__ float tabSM[];

    // Reduction intra thread
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int TID_LOCAL = Indice2D::tidLocal();

    int s = TID;
    float sum = 0.0;
    float xi;

    while (s < n)
	{
	xi = s / (float) n;
	sum += 4 / (1 + xi * xi);
	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = sum;

    __syncthreads();

    // Reduction intra block et interblock
    reductionADD(tabSM, ptrResultGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
