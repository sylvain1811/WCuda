#include "hip/hip_runtime.h"
#include "Device.h"
#include "Montecarlo.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* tabDevGeneratorGM, int n, int* ptrResultGM);
extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(int n, const Grid& grid)
    {
    this->n = n;
    this->grid = grid;
    this->pi = 0.0;
    this->sizeTabGenerator = grid.threadCounts() * sizeof(hiprandState);

    // MM
    Device::malloc(&ptrResultGM, sizeof(n0));
    Device::memclear(ptrResultGM, sizeof(n0));
    Device::malloc(&tabDevGeneratorGM, sizeTabGenerator);

    // Init generator
    int deviceId = Device::getDeviceId();
createGenerator<<<grid.dg, grid.db>>>(tabDevGeneratorGM, deviceId);

}

Montecarlo::~Montecarlo()
{
Device::free(tabDevGeneratorGM);
Device::free(ptrResultGM);
}

void Montecarlo::run()
{
int nPerThread = n / grid.threadCounts();
montecarlo<<<grid.dg, grid.db, 1024*sizeof(int)>>>(tabDevGeneratorGM, nPerThread, ptrResultGM);

Device::memcpyDToH(&n0, ptrResultGM, sizeof(int));
pi = 2 * 2 * 2 * n0 / (float) (nPerThread * grid.threadCounts());
}

int Montecarlo::getN0()
{
return this->n0;
}

float Montecarlo::getPi()
{
return this->pi;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

