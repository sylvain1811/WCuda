#include "hip/hip_runtime.h"
#include "Device.h"
#include "Montecarlo.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* tabDevGeneratorGM, int n, float* ptrResultGM);
extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(int n, const Grid& grid)
    {
    this->n = n;
    this->grid = grid;
    this->pi = 0.0;
    this->sizeTabGenerator = grid.threadCounts() * sizeof(hiprandState);

    // MM
    Device::malloc(&ptrResultGM, sizeof(pi));
    Device::memclear(ptrResultGM, sizeof(pi));
    Device::malloc(&tabDevGeneratorGM, sizeTabGenerator);

    // Init generator
    int deviceId = Device::getDeviceId();
createGenerator<<<grid.dg, grid.db>>>(tabDevGeneratorGM, deviceId);

}

Montecarlo::~Montecarlo()
{
Device::free(tabDevGeneratorGM);
Device::free(ptrResultGM);
}

void Montecarlo::run()
{
int nPerThread = n / grid.threadCounts();
montecarlo<<<grid.dg, grid.db, 1024*sizeof(float)>>>(tabDevGeneratorGM, nPerThread, ptrResultGM);

Device::memcpyDToH(&pi, ptrResultGM, sizeof(float));
pi = 2 * 4 * pi / (float) n;
}

float Montecarlo::getPi()
{
return this->pi;
}
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

