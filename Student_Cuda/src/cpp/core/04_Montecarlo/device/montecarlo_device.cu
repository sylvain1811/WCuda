#include "hip/hip_runtime.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include "Indice1D.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* tabDevGeneratorGM, int n, int* ptrResultGM);
extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    // Customisation du generator:
    // Proposition, au lecteur de faire mieux !
    // Contrainte : Doit etre différent d'un GPU à l'autre
    // Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }

__global__ void montecarlo(hiprandState* tabDevGeneratorGM, int n, int* ptrResultGM)
    {

    extern __shared__ int tabSM[];

    // Reduction intra thread
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    hiprandState localGenerator = tabDevGeneratorGM[TID];

    int n0 = 0;

    float xAlea;
    float yAlea;

    for (long i = 1; i <= n; i++)
	{
	xAlea = hiprand_uniform(&localGenerator) * 2.0 - 1.0;
	yAlea = hiprand_uniform(&localGenerator) * 2.0;

	if (sqrt(1 - xAlea * xAlea) > yAlea)
	    {
	    n0++;
	    }
	}

    tabDevGeneratorGM[TID] = localGenerator;

    tabSM[TID_LOCAL] = n0;

    __syncthreads();

    // Reduction intra block et interblock
    reductionADD(tabSM, ptrResultGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
