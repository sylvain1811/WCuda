#include "hip/hip_runtime.h"
#include "Slice.h"
#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void slice(int n, float* ptrTabGM);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Slice::Slice(int n, const Grid& grid)
    {
    this->n = n;
    this->grid = grid;
    this->sizeTab = n * sizeof(float);
    this->ptrTab = new float[n];
    this->pi = 0.0;

    // MM
    Device::malloc(&ptrTabGM, sizeTab);
    }

Slice::~Slice()
    {
    Device::free(ptrTabGM);
    delete[] ptrTab;
    }

float Slice::getPi()
    {
    return this->pi;
    }

void Slice::run()
    {
    slice<<<grid.dg, grid.db>>>(n, ptrTabGM);

    Device::memcpyDToH(ptrTab, ptrTabGM, sizeTab);

    int i = 0;
    float sum = 0.0;

#pragma omp parallel for reduction(+:sum)
    for (i = 0; i < n; i++)
	{
	sum += ptrTab[i];
	}

    pi = sum / (float) n;
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

