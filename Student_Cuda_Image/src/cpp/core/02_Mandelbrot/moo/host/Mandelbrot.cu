#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(w, h, "Mandelbrot_CUDA", domaineMath), variateurAnimation(Interval<float>(20, 120), dt)
    {
    // Input
    this->n = n;

    // Tools
    this->t = 0;

    }

Mandelbrot::~Mandelbrot(void)
    {
    // Rien
    }

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Mandelbrot::process(uchar4* ptrTabPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    mandelbrot<<<dg,db>>>(ptrDevPixels, w,h,t);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

