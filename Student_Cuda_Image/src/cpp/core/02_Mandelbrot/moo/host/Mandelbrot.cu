#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
using gpu::DomaineMath;
using namespace gpu;
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_CUDA", domaineMath), variateurAnimation(Interval<float>(20, 120), dt)
    {
    // Input
    this->n = n;

    // Tools
    this->t = 0;

    }

Mandelbrot::~Mandelbrot(void)
    {
    // Rien
    }

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("Mandelbrot rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    mandelbrot<<<dg,db>>>(ptrDevPixels, w,h,t, n, domaineMath);

    Device::lastCudaError("Mandelbrot rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

