#include "hip/hip_runtime.h"
#include <cudaTools.h>
#include <DomaineMath_GPU.h>
#include <Indice2D.h>
#include <IndiceTools_GPU.h>

#include "math/MandelbrotMath.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, uint n, DomaineMath domaineMath);
__global__ void workPixel(uchar4* ptrColorIJ, int i, int j, DomaineMath domaineMath, MandelbrotMath* ptrMandelbrotMath, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, uint n, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath = MandelbrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    // TODO pattern entrelacement
    int i;
    int j;

    int s = TID;

    while (s < WH)
	{

	IndiceTools::toIJ(s, w, &i, &j);

	//workPixel<<<>>>(&ptrDevPixels[s], i, j, domaineMath, &mandelbrotMath, t);

	double x;
	double y;
	domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

	(&mandelbrotMath)->colorXY(&ptrDevPixels[s], x, y, t); // in [01]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, MandelbrotMath* ptrMandelbrotMath, float t)
    {
    // (i,j) domaine ecran dans N2
    // (x,y) domaine math dans R2

    double x;
    double y;
    domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

    ptrMandelbrotMath->colorXY(ptrColorIJ, x, y, t); // in [01]
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

