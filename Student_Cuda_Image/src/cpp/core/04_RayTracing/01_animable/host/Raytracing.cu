#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrDevPixels,uint w, uint h,float t,int nbSphere, Sphere* ptrDevTabSphere);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    n = 400;

    assert(w == h);

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0.0f; // protected dans Animable

    sphereCreator = new SphereCreator(n, w, h);
    ptrTabSphere = sphereCreator->getTabSphere();

    sizeOctet = n * sizeof(Sphere);

    // Alloc
    Device::malloc(&ptrDevTabSphere, sizeOctet);

    // Copy
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
    }

Raytracing::~Raytracing()
    {
    // Free
    Device::free(ptrDevTabSphere);
    delete sphereCreator;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("Raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    raytracing<<<dg,db>>>(ptrDevPixels, w,h,t, n, ptrDevTabSphere);

    Device::lastCudaError("Raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

