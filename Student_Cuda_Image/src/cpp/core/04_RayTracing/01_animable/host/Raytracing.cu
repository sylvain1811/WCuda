#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "nbSphere.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrDevPixels,uint w, uint h,float t,int nbSphere, Sphere* ptrDevTabSphere);
extern __global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t);
extern __host__ void uploadToGPU(Sphere* ptrTabSphere, size_t sizeOctet);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    n = NB_SPHERE;

    assert(w == h);

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0.0f; // protected dans Animable

    sphereCreator = new SphereCreator(n, w, h);
    ptrTabSphere = sphereCreator->getTabSphere();

    sizeOctet = n * sizeof(Sphere);

    // Global memory
	{
	// Alloc
	Device::malloc(&ptrDevTabSphere, sizeOctet);

	// Copy
	Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
	}

    // Constant memory
	{
	uploadToGPU(ptrTabSphere, sizeOctet);
	}
    }

Raytracing::~Raytracing()
    {
    // Free
    Device::free(ptrDevTabSphere);
    delete sphereCreator;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("Raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    //raytracing<<<dg,db>>>(ptrDevPixels, w,h,t, n, ptrDevTabSphere);
    raytracing<<<dg,db>>>(ptrDevPixels, w,h,t);

    Device::lastCudaError("Raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

