#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "nbSphere.h"

#include "RaytracingMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_SPHERE_CM[NB_SPHERE];

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere); // GM
__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t); // CM
__host__ void uploadToGPU(Sphere* ptrTabSphere, size_t sizeOctet);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

// SM
__device__ void copyGMtoSM(Sphere* tabSM, Sphere* tabGM, int n)
    {
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    const int TID_LOCAL = Indice2D::tidLocal();
    int s = TID_LOCAL;
    while (s < n)
	{
	tabSM[s] = tabGM[s];
	s += NB_THREAD_LOCAL;
	}
    }

// CM
__host__ void uploadToGPU(Sphere* ptrTabSphere, size_t sizeOctet)
    {
    //Device::memcpyToCM(TAB_SPHERE_CM, ptrTabSphere, sizeOctet);
    hipMemcpyToSymbol(HIP_SYMBOL(TAB_SPHERE_CM), ptrTabSphere, sizeOctet, 0, hipMemcpyHostToDevice);
    }

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t) // CM
    {
    work(ptrDevPixels, w, h, t, NB_SPHERE, TAB_SPHERE_CM);
    }

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere) // GM & SM
    {
    //GM
	{
	//work(ptrDevPixels, w, h, t, nbSphere, ptrDevTabSphere); // GM
	}
    // SM
	{
	__shared__ extern Sphere tabSM[];
	copyGMtoSM(tabSM, ptrDevTabSphere, nbSphere);
	__syncthreads();
	work(ptrDevPixels, w, h, t, nbSphere, tabSM); // SM
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere)
    {
    RaytracingMath raytracingMath = RaytracingMath(w, h, nbSphere, ptrDevTabSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    // Pattern entrelacement
    int i;
    int j;

    int s = TID;

    while (s < WH)
	{

	IndiceTools::toIJ(s, w, &i, &j);

	raytracingMath.colorIJ(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

